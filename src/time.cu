#include "../include/time.cuh"
// #include <hip/hip_runtime.h>

struct GpuTime
{
    hipEvent_t start, stop;

    GpuTime()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTime()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void start()
    {
        hipEventRecord(start, 0);
    }

    void stop()
    {
        hipEventRecord(stop, 0);
    }

    double elapsed()
    {
        double elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};