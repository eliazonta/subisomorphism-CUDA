#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h> // -lcublas
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "utils/utils.cuh"
#include "parsing/parser.h"
#include "timer/time.cuh"

// thrust would be interesting asf but getting some allocation problems
// 
// #define USE_STRUCT
// #define THRUST

// idea
// kernel for subgraph isomorphism (backtracking)
// device func called by the kernel for isomorphism on a restriction of the graph 


#define BLK_SIZE 256 
#define DBG_CHECK { printf("DBG_CHECK: file %s at line %d\n", __FILE__, __LINE__ ); }
#define DEBUG

// __device__
// void isomorphism(int* g1_num_rows, int* g1_num_cols, int* g1_nnz, int* g1_rowidx, int* g1_colidx, int* g1_weights)
// {

// }

// __global__ 
// void subgraphIsomorphismKernel(struct* COOGraph g1, struct* COOGraph g2, int* mapping, bool* found, int patternVertices, int targetVertices) 
// {
//     unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     if (tid >= patternVertices) {
//         return;  // not enough threads for all pattern vertices
//     }

//     if (*found) {
//         return;  // solution already found by another thread
//     }

//     // implement the backtracking logic for this thread!!
//     // explore different branches of the search space and update the mapping

//     // if a valid solution is found, set *found = true
// }

int main(int argc, char** argv) {
    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(1, 1, 1);
    char* file = "../toy/g2.mtx";

    #ifdef USE_STRUCT
    struct COOGraph g1, g2;
    struct COOGraph d_g1, d_g2;
    readCOO_struct(file, &g1);
    readCOO_struct(file, &g2);
    #endif

    #ifndef USE_STRUCT
        int g1_num_edges, g2_num_edges;
        // g1 - noTHRUST
        #ifndef THRUST
        int* g1_rowidx, g1_colidx, d_g1_rowidx, d_g1_colidx;
        dtype* g1_weights, d_g1_weights;
        #endif

        // g1 - THRUST 
        #ifdef THRUST
        thrust::host_vector<int> g1_rowidx, g1_colidx;
        thrust::host_vector<dtype> g1_weights;
        #endif


        // g2 - noTHRUST
        #ifndef THRUST
        int* g2_rowidx, g2_colidx, d_g2_rowidx, d_g2_colidx;
        dtype* g2_weights, d_g2_weights;
        #endif

        // g2 - THRUST 
        #ifdef THRUST
        thrust::host_vector<int> g2_rowidx, g2_colidx;
        thrust::host_vector<dtype> g2_weights;
        #endif

        // readCOO(file, g1_num_edges, &g1_rowidx, &g1_colidx, &g1_weights);
        // readCOO(file, g2_num_edges, &g2_rowidx, &g2_colidx, &g2_weights);
    #endif

    
    // #ifdef USE_STRUCT //fix
    // readCOO_struct("../../toy/g2.mtx", &g1);
    // readCOO_struct("../../toy/g2.mtx", &g2);
    // // copy of stack data for g1
    // d_g1.num_edges = g1.num_edges;
    // d_g1.num_rows = g1.num_rows;
    // d_g1.num_cols = g1.num_cols;
    // // d_g1.rowidx = malloc_device<int>(d_g1.num_rows + 1);
    // // d_g1.colidx = malloc_device<int>(d_g1.num_cols + 1);

    // // copy of stack data for g2
    // d_g2.nnz = g2.nnz;
    // d_g2.num_rows = g2.num_rows;
    // d_g2.num_cols = g2.num_cols;
    // // d_g2.rowidx = malloc_device<int>(d_g2.num_rows + 1);
    // // d_g2.colidx = malloc_device<int>(d_g2.num_cols + 1);
    // #endif

    
    // GpuTime time_H2D;
    // time_H2D.start();

    // #ifdef USE_STRUCT
    // // copy to device for g1
    // copy_to_device(d_g1.rowidx, g1.rowidx, (g1.num_edges + 1) * sizeof(int));
    // copy_to_device(d_g1.colidx, g1.colidx, (g1.num_edges + 1) * sizeof(int));
    // // thrust::device_vector<int> d_g1.rowidx = g1.rowidx; // has to be fixed, even just for fun :)
    // // thrust::device_vector<int> d_g1.colidx = g1.colidx;

    // // copy to device for g2
    // copy_to_device(d_g2.rowidx, g2.rowidx, (g2.num_edges + 1) * sizeof(int));
    // copy_to_device(d_g2.colidx, g2.colidx, (g2.num_edges + 1) * sizeof(int));
    // // thrust::device_vector<int> d_g2.rowidx = g2.rowidx;
    // // thrust::device_vector<int> d_g2.colidx = g2.colidx;
    // #endif

    // #ifndef USE_STRUCT
    // // copy to device for g1
    // GpuTime timeH2D_g1;
    // timeH2D_g1.start();
    // thrust::device_vector<int> d_g1_rowidx = g1_rowidx;
    // thrust::device_vector<int> d_g1_colidx = g1_colidx;
    // thrust::device_vector<dtype> d_g1_weights = g1_weights;
    // timeH2D_g1.stop();
    // printf("G1 H2D elapsed time: %f ms", time_H2D_g1.elapsed());
    
    // // copy to device for g2
    // GpuTime timeH2D_g2;
    // timeH2D_g2.start();
    // thrust::device_vector<int> d_g2_rowidx = g2_rowidx;
    // thrust::device_vector<int> d_g2_colidx = g2_colidx;
    // thrust::device_vector<dtype> d_g2_weights = g2_weights;
    // timeH2D_g2.stop();
    // printf("G2 H2D elapsed time: %f ms", time_H2D_g2.elapsed());
    // #endif
    // // elapsed
    // timeH2D.stop();
    // printf("Complete H2D elapsed time: %f ms", time_H2D.elapsed());

    // // CUDA kernel 
    // // Call subgraphIsomorphismKernel<<<gridSize, blockSize>>>(targetGraph, patternGraph, mapping, found, patternVertices, targetVertices);
    // // coolSubgraphIsomorhismKernel<<<grid_size, block_size>>>(targetG, patternG, mapping, ....);
    
    // // Wait for the kernel to finish
    // // cuda_check_status(cudaDeviceSyncronize());

    // // Copy the results (e.g., valid mappings) from GPU to CPU
    // #ifdef USE_STRUCT
    // #endif

    // #ifndef USE_STRUCT
    // copy_to_host(d_g1_rowidx, g1_rowidx);
    // copy_to_host(d_g1_colidx, g1_colidx);
    // copy_to_host(d_g1_weights, g1_weights);

    // copy_to_host(d_g2_rowidx, g2_rowidx);
    // copy_to_host(d_g2_colidx, g2_colidx);
    // copy_to_host(d_g2_weights, g2_weights);

    // #endif

    // #ifdef USE_STRUCT
    // // host
    // freeCOO_struct(&g1);
    // freeCOO_struct(&g2);
    
    // // device
    // hipFree(d_g1.rowidx);
    // hipFree(d_g1.colidx);
    // hipFree(d_g1.weights);

    // hipFree(d_g2.rowidx);
    // hipFree(d_g2.colidx);
    // hipFree(d_g2.weights);
    // #endif

    // #ifndef USE_STRUCT
    // // host
    // free(g1_rowidx);
    // free(g1_colidx);
    // free(g1_weights);

    // free(g2_rowidx);
    // free(g2_colidx);
    // free(g2_weights);

    // // device
    // hipFree(d_g1_rowidx);
    // hipFree(d_g1_colidx);
    // hipFree(d_g1_weights);

    // hipFree(d_g2_rowidx);
    // hipFree(d_g2_colidx);
    // hipFree(d_g2_weights);
    // #endif

    return 0;
}
