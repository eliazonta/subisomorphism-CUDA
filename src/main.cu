#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "../include/GPU_TIME.h"
#include "../include/parser.h"

/*
    IDEA:
        - CSR parser 
        - Sequential alg on CSR vectors
        - design a paralell GPU alg
        - benchmark and tests
*/
__global__ 
void gpu_print(void){
    printf("Hello from thread [%d, %d]\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char** argv)
{   
    if (argc != 5) {
        fprintf(stdout, "Invalid command, enter:\n1. number of threads, 2. number of repetitions, 3. print mode (1 or 2), 4. test filename\n");
        exit(-1);
    }
    const char *filename = argv[2];
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals, numSMs;
    float* values;
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    return 0;
}