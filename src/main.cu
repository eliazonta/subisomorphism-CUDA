#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include "../include/iso.h"
#include "../include/utils.cuh"
#include "../include/parser.h"

// #define USE_STRUCT
__device__
void isomorphism(struct COOGraph g1, struct COOGraph g2){

}

__global__ 
void subgraphIsomorphismKernel(int* targetGraph, int* patternGraph, int* mapping, bool* found, int patternVertices, int targetVertices) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= patternVertices) {
        return;  // Not enough threads for all pattern vertices
    }

    if (*found) {
        return;  // Solution already found by another thread
    }

    // Implement the backtracking logic for this thread
    // Explore different branches of the search space and update the mapping

    // If a valid solution is found, set *found = true
}

int main(int argc, char** argv) {
    char file[] = "../toy/g2.mtx";

    #ifdef USE_STRUCT
    struct COOGraph g1, g2;
    struct COOGraph d_g1, d_g2;
    readCOO_struct(file, &g1);
    readCOO_struct(file, &g2);
    #endif

    #ifdef NO_STRUCT
        int num_rows, num_cols, nnz;
        int* rowidx, colidx;
        double* weights;
    #endif
    // Initialize and allocate GPU memory for targetGraph, patternGraph, mapping, and other necessary data
    
    // copy of stack data for g1
    d_g1.nnz = g1.nnz;
    d_g1.num_rows = g1.num_rows;
    d_g1.num_cols = g1.num_cols;
    // d_g1.rowidx = malloc_device<int>(d_g1.num_rows + 1);
    // d_g1.colidx = malloc_device<int>(d_g1.num_cols + 1);

    // copy of stack data for g2
    d_g2.nnz = g2.nnz;
    d_g2.num_rows = g2.num_rows;
    d_g2.num_cols = g2.num_cols;
    // d_g2.rowidx = malloc_device<int>(d_g2.num_rows + 1);
    // d_g2.colidx = malloc_device<int>(d_g2.num_cols + 1);

    
    auto s = get_time();
    
    // copy to device for g1
    // copy_to_device(d_g1.rowidx, g1.rowidx, (g1.num_rows + 1) * sizeof(int));
    // copy_to_device(d_g1.colidx, g1.colidx, (g1.num_cols + 1) * sizeof(int));
    thrust::device_vector<int> d_g1.rowidx = g1.rowidx;
    thrust::device_vector<int> d_g1.colidx = g1.colidx;
    // copy to device for g2
    // copy_to_device(d_g2.rowidx, g2.rowidx, (g2.num_rows + 1) * sizeof(int));
    // copy_to_device(d_g2.colidx, g2.colidx, (g2.num_cols + 1) * sizeof(int));
    thrust::device_vector<int> d_g2.rowidx = g2.rowidx;
    thrust::device_vector<int> d_g2.colidx = g2.colidx;
    
    // elapsed
    auto time_H2D = get_time() - s;

    // Launch the CUDA kernel with appropriate grid and block sizes
    // Call subgraphIsomorphismKernel<<<gridSize, blockSize>>>(targetGraph, patternGraph, mapping, found, patternVertices, targetVertices);

    // Wait for the kernel to finish

    // Copy the results (e.g., valid mappings) from GPU to CPU
    

    // Free GPU memory and clean up
    hipFree(d_g1.rowidx);
    hipFree(d_g1.colidx);

    hipFree(d_g2.rowidx);
    hipFree(d_g2.colidx);

    return 0;
}
