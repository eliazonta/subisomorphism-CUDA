#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include "../include/iso.h"
#include "../include/utils.cuh"
#include "../include/parser.h"

// #define USE_STRUCT

// idea
// kernel for subgraph isomorphism (backtracking)
// device func called by the kernel for isomorphism on a restriction of the graph 

__device__
void isomorphism(int* g1_num_rows, int* g1_num_cols, int* g1_nnz, int* g1_rowidx, int* g1_colidx, int* g1_weights){

}

__global__ 
void subgraphIsomorphismKernel(int* targetGraph, int* patternGraph, int* mapping, bool* found, int patternVertices, int targetVertices) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= patternVertices) {
        return;  // Not enough threads for all pattern vertices
    }

    if (*found) {
        return;  // Solution already found by another thread
    }

    // Implement the backtracking logic for this thread
    // Explore different branches of the search space and update the mapping

    // If a valid solution is found, set *found = true
}

int main(int argc, char** argv) {
    char file[] = "../toy/g2.mtx";

    #ifdef USE_STRUCT
    struct COOGraph g1, g2;
    struct COOGraph d_g1, d_g2;
    readCOO_struct(file, &g1);
    readCOO_struct(file, &g2);
    #endif

    #ifndef USE_STRUCT
        // g1
        int g1_num_rows, g1_num_cols, g1_nnz;
        int* g1_rowidx, g1_colidx, g1_d_row_idx, g1_d_rowidx, g1_d_colidx;
        double* g1_weights, g1_d_weights;

        // g1
        int g2_num_rows, g2_num_cols, g2_nnz;
        int* g2_rowidx, g2_colidx, g2_d_row_idx, g2_d_rowidx, g2_d_colidx;
        double* g2_weights, g2_d_weights;

        readCOO(file, &g1_num_rows, &g1_num_cols, &g1_nnz, &g1_rowidx, &g1_colidx, &g1_weights);
        readCOO(file, &g2_num_rows, &g2_num_cols, &g2_nnz, &g2_rowidx, &g2_colidx, &g2_weights);
    #endif
    // Initialize and allocate GPU memory for targetGraph, patternGraph, mapping, and other necessary data
    #ifdef USE_STRUCT
    // copy of stack data for g1
    d_g1.nnz = g1.nnz;
    d_g1.num_rows = g1.num_rows;
    d_g1.num_cols = g1.num_cols;
    // d_g1.rowidx = malloc_device<int>(d_g1.num_rows + 1);
    // d_g1.colidx = malloc_device<int>(d_g1.num_cols + 1);

    // copy of stack data for g2
    d_g2.nnz = g2.nnz;
    d_g2.num_rows = g2.num_rows;
    d_g2.num_cols = g2.num_cols;
    // d_g2.rowidx = malloc_device<int>(d_g2.num_rows + 1);
    // d_g2.colidx = malloc_device<int>(d_g2.num_cols + 1);
    #endif

    
    auto s = get_time();

    #ifdef USE_STRUCT
    // copy to device for g1
    copy_to_device(d_g1.rowidx, g1.rowidx, (g1.num_rows + 1) * sizeof(int));
    copy_to_device(d_g1.colidx, g1.colidx, (g1.num_cols + 1) * sizeof(int));
    // thrust::device_vector<int> d_g1.rowidx = g1.rowidx; // has to be fixed, even just for fun :)
    // thrust::device_vector<int> d_g1.colidx = g1.colidx;
    // copy to device for g2
    copy_to_device(d_g2.rowidx, g2.rowidx, (g2.num_rows + 1) * sizeof(int));
    copy_to_device(d_g2.colidx, g2.colidx, (g2.num_cols + 1) * sizeof(int));
    // thrust::device_vector<int> d_g2.rowidx = g2.rowidx;
    // thrust::device_vector<int> d_g2.colidx = g2.colidx;
    #endif

    #ifndef USE_STRUCT
    // copy to device for g1
    thrust::device_vector<int> g1_d_rowidx = g1_rowidx;
    thrust::device_vector<int> g1_d_colidx = g1_colidx;
    thrust::device_vector<double> g1_d_weights = g1_weights;

    // copy to device for g2
    thrust::device_vector<int> g2_d_rowidx = g2_rowidx;
    thrust::device_vector<int> g2_d_colidx = g2_colidx;
    thrust::device_vector<double> g2_d_weights = g2_weights;
        
    #endif
    // elapsed
    auto time_H2D = get_time() - s;

    // Launch the CUDA kernel with appropriate grid and block sizes
    // Call subgraphIsomorphismKernel<<<gridSize, blockSize>>>(targetGraph, patternGraph, mapping, found, patternVertices, targetVertices);

    // Wait for the kernel to finish

    // Copy the results (e.g., valid mappings) from GPU to CPU
    
    #ifdef USE_STRUCT
    // Free GPU memory and clean up
    hipFree(d_g1.rowidx);
    hipFree(d_g1.colidx);

    hipFree(d_g2.rowidx);
    hipFree(d_g2.colidx);
    #endif

    #ifndef USE_STRUCT
    hipFree(g1_d_rowidx);
    hipFree(g1_d_colidx);

    hipFree(g2_d_rowidx);
    hipFree(g2_d_colidx);
    #endif

    return 0;
}
