#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
// #include "../include/iso.h"
#include "../include/utils.cuh"
#include "../include/parser.h"
#include "../include/time.cuh"

// #define USE_STRUCT
#define THRUST

// idea
// kernel for subgraph isomorphism (backtracking)
// device func called by the kernel for isomorphism on a restriction of the graph 


#define BLK_SIZE 256
#define dtype float 
#define DEBUG

__device__
void isomorphism(int* g1_num_rows, int* g1_num_cols, int* g1_nnz, int* g1_rowidx, int* g1_colidx, int* g1_weights)
{

}

__global__ 
void subgraphIsomorphismKernel(int* targetGraph, int* patternGraph, int* mapping, bool* found, int patternVertices, int targetVertices) 
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= patternVertices) {
        return;  // Not enough threads for all pattern vertices
    }

    if (*found) {
        return;  // Solution already found by another thread
    }

    // Implement the backtracking logic for this thread
    // Explore different branches of the search space and update the mapping

    // If a valid solution is found, set *found = true
}

int main(int argc, char** argv) {
    dim3 block_size(BLK_SIZE, 1, 1);
    dim3 grid_size(1,1 ,1);
    char file[] = "../toy/g2.mtx";

    #ifdef USE_STRUCT
    struct COOGraph g1, g2;
    struct COOGraph d_g1, d_g2;
    readCOO_struct(file, &g1);
    readCOO_struct(file, &g2);
    #endif

    #ifndef USE_STRUCT
        // g1
        #ifndef THRUST
        int g1_num_rows, g1_num_cols, g1_nnz;
        int* g1_rowidx, g1_colidx, g1_d_row_idx, g1_d_rowidx, g1_d_colidx;
        double* g1_weights, g1_d_weights;
        #endif

        // better g1 but wip
        #ifdef THRUST
        thrust::host_vector<int> g1_nnz, g1_num_edges, g1_rowidx, g1_colidx;
        thrust::host_vector<double> g1_weights;
        
        #endif


        // g2
        #ifndef THRUST
        int g2_num_rows, g2_num_cols, g2_nnz;
        int* g2_rowidx, g2_colidx, g2_d_row_idx, g2_d_rowidx, g2_d_colidx;
        double* g2_weights, g2_d_weights;
        #endif

        // better g2 but wip
        #ifdef THRUST
        thrust::host_vector<int> g2_nnz, g2_num_edges, g2_rowidx, g2_colidx;
        thrust::host_vector<double> g2_weights;
        thrust::device_vector<int> d_g2_nnz, d_g2_num_edges, d_g2_rowidx, d_g2_colidx;
        thrust::device_vector<double> d_g2_weights;
        #endif

        readCOO(file, &g1_num_rows, &g1_num_cols, &g1_nnz, &g1_rowidx, &g1_colidx, &g1_weights);
        readCOO(file, &g2_num_rows, &g2_num_cols, &g2_nnz, &g2_rowidx, &g2_colidx, &g2_weights);
    #endif

    // Initialize and allocate GPU memory for targetGraph, patternGraph, mapping, and other necessary data
    #ifdef USE_STRUCT
    // copy of stack data for g1
    d_g1.nnz = g1.nnz;
    d_g1.num_rows = g1.num_rows;
    d_g1.num_cols = g1.num_cols;
    // d_g1.rowidx = malloc_device<int>(d_g1.num_rows + 1);
    // d_g1.colidx = malloc_device<int>(d_g1.num_cols + 1);

    // copy of stack data for g2
    d_g2.nnz = g2.nnz;
    d_g2.num_rows = g2.num_rows;
    d_g2.num_cols = g2.num_cols;
    // d_g2.rowidx = malloc_device<int>(d_g2.num_rows + 1);
    // d_g2.colidx = malloc_device<int>(d_g2.num_cols + 1);
    #endif

    
    auto s = get_time();

    #ifdef USE_STRUCT
    // copy to device for g1
    copy_to_device(d_g1.rowidx, g1.rowidx, (g1.num_rows + 1) * sizeof(int));
    copy_to_device(d_g1.colidx, g1.colidx, (g1.num_cols + 1) * sizeof(int));
    // thrust::device_vector<int> d_g1.rowidx = g1.rowidx; // has to be fixed, even just for fun :)
    // thrust::device_vector<int> d_g1.colidx = g1.colidx;

    // copy to device for g2
    copy_to_device(d_g2.rowidx, g2.rowidx, (g2.num_rows + 1) * sizeof(int));
    copy_to_device(d_g2.colidx, g2.colidx, (g2.num_cols + 1) * sizeof(int));
    // thrust::device_vector<int> d_g2.rowidx = g2.rowidx;
    // thrust::device_vector<int> d_g2.colidx = g2.colidx;
    #endif

    #ifndef USE_STRUCT
    // copy to device for g1
    GpuTime timeH2D_g1;
    timeH2D_g1.start();
    thrust::device_vector<int> d_g1_nnz = g1_nnz;
    thrust::device_vector<int> d_g1_num_edges = g1_num_edges;
    thrust::device_vector<int> d_g1_rowidx = g1_rowidx;
    thrust::device_vector<int> d_g1_colidx = g1_colidx;

    thrust::device_vector<double> d_g1_weights = g1_weights;
    timeH2D_g1.stop();
    printf("G1 H2D elapsed time: %f ms", time_H2D_g1.elapsed());
    
    // copy to device for g2
    thrust::device_vector<int> d_g2_nnz = g2_nnz;
    thrust::device_vector<int> d_g2_num_edges = g2_num_edges;
    thrust::device_vector<int> d_g2_rowidx = g2_rowidx;
    thrust::device_vector<int> d_g2_colidx = g2_colidx;
        
    #endif
    // elapsed
    auto time_H2D = get_time() - s;

    // CUDA kernel 
    // Call subgraphIsomorphismKernel<<<gridSize, blockSize>>>(targetGraph, patternGraph, mapping, found, patternVertices, targetVertices);
    // coolSubgraphIsomorhismKernel<<<grid_size, block_size>>>(targetG, patternG, mapping, ....);
    
    // Wait for the kernel to finish
    // cuda_check_status(cudaDeviceSyncronize());

    // Copy the results (e.g., valid mappings) from GPU to CPU
    #ifdef USE_STRUCT
    #endif

    #ifndef USE_STRUCT
    copy_to_host(g1_d_rowidx, g1_rowidx);
    copy_to_host(g1_d_colidx, g1_colidx);
    copy_to_host(g1_d_weights, g1_weights);

    copy_to_host(g2_d_rowidx, g2_rowidx);
    copy_to_host(g2_d_colidx, g2_colidx);
    copy_to_host(g2_d_weights, g2_weights);

    #endif

    #ifdef USE_STRUCT
    // Free GPU memory and clean up
    hipFree(d_g1.rowidx);
    hipFree(d_g1.colidx);

    hipFree(d_g2.rowidx);
    hipFree(d_g2.colidx);
    #endif

    #ifndef USE_STRUCT
    hipFree(g1_d_rowidx);
    hipFree(g1_d_colidx);

    hipFree(g2_d_rowidx);
    hipFree(g2_d_colidx);
    #endif

    return 0;
}
