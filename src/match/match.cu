#include "hip/hip_runtime.h"

#include "match.cuh"

// gpu warmup needs to be done otherwise first malloc will be slower
void initGPU(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    hipSetDevice(dev);
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %luB; compute v%d.%d; clock: %d kHz; shared mem: %dB; block threads: %d; SM count: %d\n",
               devProps.name, devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate,
			   devProps.sharedMemPerBlock, devProps.maxThreadsPerBlock, devProps.multiProcessorCount);
    }
	fprintf(stdout, "on GPU!\n");
	// GPU init
	int* warmup = NULL;
	hipMalloc(&warmup, sizeof(int));
	hipFree(warmup);
	fprintf(stdout, "GPU warmup done\n");
    size_t size = 0x7fffffff;   //ulong , appr 2G
    size *= 4;
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	fprintf(stdout, "check heap limit: %ul\n", size);
}




DEVICE_INTRINSIC_QUALIFIERS
unsigned int
smid()
{
  unsigned int r;
  asm("mov.u32 %0, %%smid;" : "=r"(r));
  return r;
}

DEVICE_INTRINSIC_QUALIFIERS
unsigned int
nsmid()
{
#if (__CUDA_ARCH__ >= 200)
  unsigned int r;
  asm("mov.u32 %0, %%nsmid;" : "=r"(r));
  return r;
#else
  return 30;
#endif
}

void exclusive_sum(unsigned* d_array, unsigned size)
{
    // Determine temporary device storage requirements
    void* d_temp_storage = NULL; //must be set to distinguish two phase
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_array, d_array, size);
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_array, d_array, size);
    hipFree(d_temp_storage);
}