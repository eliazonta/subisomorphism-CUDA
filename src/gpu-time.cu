#include "../include/gpu-time.h"

struct GpuTime
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTime()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTime()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void start()
    {
        hipEventRecord(start, 0);
    }

    void stop()
    {
        hipEventRecord(stop, 0);
    }

    double elapsed()
    {
        double elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};