#include "hip/hip_runtime.h"
#include "../include/iso.h"
#inclide "../include/utils.cuh"

__device__
void isomorphism(struct COOGraph g1, struct COOGraph g2){

}

__global__ 
void subgraphIsomorphismKernel(int* targetGraph, int* patternGraph, int* mapping, bool* found, int patternVertices, int targetVertices) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= patternVertices) {
        return;  // Not enough threads for all pattern vertices
    }

    if (*found) {
        return;  // Solution already found by another thread
    }

    // Implement the backtracking logic for this thread
    // Explore different branches of the search space and update the mapping

    // If a valid solution is found, set *found = true
}

int main() {
    char file[] = "../toy/g2.mtx";
    struct COOGraph g1, g2;
    struct COOGraph d_g1, d_g2;
    readCOO(file, &g1);
    readCOO(file, &g2);
    // Initialize and allocate GPU memory for targetGraph, patternGraph, mapping, and other necessary data
    d_g1.num_vertices = g1.num_vertices;
    d_g1.num_edges = g1.num_edges;
    d_g1.row_ptr = malloc_device<int>();
    d_g1.col_idx = malloc_device<int>();

    d_g2.num_vertices = g2.num_vertices;
    d_g2.num_edges = g2.num_edges;
    d_g2.row_ptr = malloc_device<int>();
    d_g2.col_idx = malloc_device<int>();
    // Launch the CUDA kernel with appropriate grid and block sizes
    // Call subgraphIsomorphismKernel<<<gridSize, blockSize>>>(targetGraph, patternGraph, mapping, found, patternVertices, targetVertices);

    // Wait for the kernel to finish

    // Copy the results (e.g., valid mappings) from GPU to CPU
    

    // Free GPU memory and clean up
    hipFree(d_g1.row_ptr);
    hipFree(d_g1.col_idx);

    hipFree(d_g2.row_ptr);
    hipFree(d_g2.col_idx);

    return 0;
}
