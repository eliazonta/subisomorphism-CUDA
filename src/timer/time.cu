#include "../include/time.cuh"
// #include <hip/hip_runtime.h>

struct GpuTime
{
    hipEvent_t sstart, sstop;

    GpuTime()
    {
        hipEventCreate(&sstart);
        hipEventCreate(&sstop);
    }

    ~GpuTime()
    {
        hipEventDestroy(sstart);
        hipEventDestroy(sstop);
    }

    void start(int stream = 0)
    {
        hipEventRecord(sstart, stream);
    }

    void stop(int stream = 0)
    {
        hipEventRecord(sstop, stream);
    }

    double elapsed()
    {
        double elapsed;
        hipEventSynchronize(sstop);
        hipEventElapsedTime(&elapsed, sstart, sstop);
        return elapsed;
    }
};